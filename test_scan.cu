﻿/*
* test_scan.cu
*
* Файл содержит реализацию следующих функций:
* test_scan
*
* Автор: Гоглачев Андрей Игоревич, ЮУрГУ, 2021 год
*/

#include <hip/hip_runtime.h>
#include <>

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include "scan.cuh"

void test_scan() {
	int n = 1 << 7;
	int bytes = n * sizeof(int);

	int numThreads = 1 << 5;
	int numBlocks = (n + (numThreads * 2 - 1)) / (numThreads * 2);
	int smemSize = numThreads * sizeof(int) * 2;

	int* h_idata = (int*)malloc(bytes);
	assert(h_idata != NULL);
	int* h_odata = (int*)malloc(bytes);
	assert(h_odata != NULL);
	int* h_cumsum = (int*)malloc(bytes);
	assert(h_cumsum != NULL);

	int sum = 0;
	for (int i = 0; i < n; i++) {
		h_cumsum[i] = sum;
		int a = (rand() & 0xFF);
		h_idata[i] = a;
		sum += a;	
	}

	int* d_idata = NULL;
	int* d_cumsum = NULL;
	int* d_blocksum = NULL;

	hipError_t result = hipMalloc((void**)&d_idata, bytes);
	assert(result == hipSuccess);
	result = hipMalloc((void**)&d_cumsum, bytes);
	assert(result == hipSuccess);
	result = hipMalloc((void**)&d_blocksum, numBlocks * sizeof(int));
	assert(result == hipSuccess);

	result = hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
	assert(result == hipSuccess);

	scan<int> << <numBlocks, numThreads, smemSize >> > (d_idata, d_cumsum, d_blocksum, n);
	scan<int> << <1, 32, 32 * sizeof(int) >> > (d_blocksum, d_blocksum, numBlocks);
	add<int> << <numBlocks, numThreads * 2 >> > (d_cumsum, d_cumsum, d_blocksum);

	result = hipMemcpy(h_odata, d_cumsum, bytes, hipMemcpyDeviceToHost);
	assert(result == hipSuccess);

	for (int i = 0; i < n; i++) {
		assert(h_cumsum[i] == h_odata[i]);
	}

	hipFree(d_idata);
	hipFree(d_cumsum);
	hipFree(d_blocksum);

	free(h_idata);
	free(h_odata);
	free(h_cumsum);
}